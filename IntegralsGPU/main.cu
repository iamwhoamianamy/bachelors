#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include <string>
#include <cmath>
#include <iomanip>
#include <chrono>

#include "triangle_quadratures.h"
#include "mesh.h"
#include "laplace_solver.h"
#include "cuda_timer.cuh"
#include "laplace_solver_arrays.cuh"

using namespace std;
using namespace triangle_quadratures;
using namespace cuda_utilities;
namespace lscpu = laplace_solver;

int main()
{
   Mesh mesh;

   try
   {
      mesh.InitFromOBJ("../meshes/icosphere_highres.obj");
   }
   catch(Exeption fileExeption)
   {
      cout << fileExeption;
      exit(1);
   }

   QuadPoints qp;

   try
   {
      qp.InitFromTXT("../quadratures/gauss15_xy.txt", "../quadratures/gauss15_w.txt");
   }
   catch(Exeption fileExeption)
   {
      cout << fileExeption;
      exit(1);
   }

   Vector3 n;

   const int points_count = 10;
   vector<double> res;
   vector<Vector3> points(points_count);

   for(size_t i = 0; i < points_count; i++)
   {
      points[i] = { 0.8 / points_count * (i + 1), 0.20, 0.00 };
   }


   hipDeviceProp_t dev_prop;
   int device_count;
   hipGetDeviceCount(&device_count);

   if(!device_count)
   {
      cout << "No cuda compatable devices found!" << endl;
      exit(2);
   }

   //// GPU
   //cout << "GPU computation:" << endl;
   //lsgpu::calcIntegralOverMesh(mesh, qp, points, res);

   //for(size_t i = 0; i < points_count; i++)
   //{
   //   cout << "Point: " << scientific << points[i].x << " " << points[i].y << " " << points[i].z << endl;

   //   double true_value = laplace_solver::u(points[i]);
   //   double calc_value = res[i];
   //   double error = abs((true_value - calc_value) / true_value);

   //   cout << "Integral:" << endl;
   //   cout << fixed;
   //   cout << "True value =       " << setw(16) << true_value << endl;
   //   cout << "Calculated value = " << setw(16) << calc_value << endl;
   //   cout << scientific;
   //   cout << "Error            = " << setw(16) << error << endl;
   //}

   //cout << endl << "-----------------------------------------------" << endl << endl;

   // CPU
   auto start = std::chrono::steady_clock::now();
   cout << "CPU computation:" << endl;

   lscpu::calcIntegralOverMesh(mesh, qp, points, res);

   auto stop = std::chrono::steady_clock::now();
   auto ellapsed_time_cpu = chrono::duration_cast<chrono::microseconds>(stop - start).count() * 1e-6;
   cout << "Calculation time: " << ellapsed_time_cpu << endl << endl;

   for(size_t i = 0; i < points_count; i++)
   {
      cout << "Point: " << scientific << points[i].x << " " << points[i].y << " " << points[i].z << endl;

      double true_value = laplace_solver::u(points[i]);
      double calc_value = res[i];
      double error = abs((true_value - calc_value) / true_value);

      cout << "Integral:" << endl;
      cout << fixed;
      cout << "True value =       " << setw(16) << true_value << endl;
      cout << "Calculated value = " << setw(16) << calc_value << endl;
      cout << scientific;
      cout << "Error            = " << setw(16) << error << endl;
   }

   cout << endl << "-----------------------------------------------" << endl << endl;

   LaplaceSolverArrays laplaceSolverArrays;
   laplaceSolverArrays.PrepareData(points, mesh, qp);
   laplaceSolverArrays.CopyToDevice();
   laplaceSolverArrays.SolveGPU();
   res = laplaceSolverArrays.GetResultGPU();

   for(size_t i = 0; i < points_count; i++)
   {
      cout << "Point: " << scientific << points[i].x << " " << points[i].y << " " << points[i].z << endl;

      double true_value = laplace_solver::u(points[i]);
      double calc_value = res[i];
      double error = abs((true_value - calc_value) / true_value);

      cout << "Integral:" << endl;
      cout << fixed;
      cout << "True value =       " << setw(16) << true_value << endl;
      cout << "Calculated value = " << setw(16) << calc_value << endl;
      cout << scientific;
      cout << "Error            = " << setw(16) << error << endl;
   }

   return 0;
}