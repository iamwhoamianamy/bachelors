#include "cuda_timer.h"

using namespace cuda_utilities;

CudaTimer::CudaTimer()
{
   hipEventCreate(&start);
   hipEventCreate(&stop);
}

CudaTimer::~CudaTimer()
{
   hipEventDestroy(start);
   hipEventDestroy(stop);
}

void CudaTimer::Start()
{
   hipEventRecord(start, 0);
}

float CudaTimer::Ellapsed()
{
   float ellapsed_time = 0;

   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&ellapsed_time, start, stop);

   return ellapsed_time * 1e-3;
}