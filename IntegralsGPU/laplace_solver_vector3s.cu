#include "laplace_solver_vector3s.h"
#include "hip/hip_runtime.h"
#include ""
#include "laplace_data.cuh"
#include "cuda_helper.h"
#include "laplace_solver_kernels.cuh"

//LaplaceSolverVector3s::LaplaceSolverVector3s(AlgorythmGPU alg) : algorythmGPU(alg) {};
LaplaceSolverVector3s::LaplaceSolverVector3s() {};

using namespace laplace_data;

void LaplaceSolverVector3s::PrepareData(vector<Vector3>& points, Mesh& mesh, BasisQuadratures& basisQuads) 
{
   quadraturesCount = basisQuads.order * mesh.TrianglesCount();
   trianglesCount = mesh.TrianglesCount();
   pointsCount = points.size();
   quadraturesOrder = basisQuads.order;

   // Preparing quadPoints
   quadPoints.resize(quadraturesCount);

   for(size_t t = 0; t < trianglesCount; t++)
   {
      Triangle tr = mesh.GetTriangle(t);

      for(size_t o = 0; o < basisQuads.order; o++)
      {
         int ind = t * basisQuads.order + o;
         quadPoints[ind] = tr.PointFromST(basisQuads.x[o], basisQuads.y[o]);
      }
   }

   // Preparing normals
   normals.resize(trianglesCount);

   for(size_t t = 0; t < trianglesCount; t++)
   {
      Triangle tr = mesh.GetTriangle(t);
      normals[t] = tr.Normal();
   }

   // Preparing points
   this->points = vector<Vector3>(points);

   // Preparing weights
   weights = vector<float>(basisQuads.w);

   // Preparing areas
   areas.resize(trianglesCount);

   for(size_t t = 0; t < trianglesCount; t++)
   {
      areas[t] = mesh.GetTriangle(t).Area();
   }

   // Preparing results
   results = vector<float>(pointsCount, 0);
}

void LaplaceSolverVector3s::CopyToDevice() 
{
   // Copying quadPoints
   dev_quadPoints = DevPtr<Vector3>(quadPoints.data(), quadraturesCount);

   // Copying normals
   dev_normals = DevPtr<Vector3>(normals.data(), trianglesCount);

   // Copying points
   dev_points = DevPtr<Vector3>(points.data(), pointsCount);

   // Copying weights
   dev_weights = DevPtr<float>(weights.data(), weights.size());

   // Copying areas
   dev_areas = DevPtr<float>(areas.data(), trianglesCount);

   // Copying results
   dev_results = DevPtr<float>(pointsCount);
}

vector<float>& LaplaceSolverVector3s::SolveCPU()
{
   for(size_t p = 0; p < pointsCount; p++)
   {
      float integral = 0;

      for(size_t t = 0; t < trianglesCount; t++)
      {
         float tringle_sum_1 = 0;
         float tringle_sum_2 = 0;

         for(size_t o = 0; o < quadraturesOrder; o++)
         {
            int ind = t * quadraturesOrder + o;
            tringle_sum_1 += weights[o] * laplaceIntegral1(quadPoints[ind],
                                                           points[p],
                                                           normals[t]);

            tringle_sum_2 += weights[o] * laplaceIntegral2(quadPoints[ind],
                                                           points[p],
                                                           normals[t]);
         }

         integral += (tringle_sum_1 - tringle_sum_2) * areas[t];
      }

      results[p] = integral / (4.0 * PI);
   }

   return results;
}

void LaplaceSolverVector3s::SolveGPU()
{
   switch(algorythmGPU)
   {
      case AlgorythmGPU::Reduction:
      {
         laplace_solver_kernels::SolverKernelVector3sReduction<<<
            pointsCount,
            THREADS_PER_BLOCK,
            THREADS_PER_BLOCK * sizeof(float)>>>(
               dev_quadPoints.Get(),
               dev_normals.Get(),
               dev_points.Get(),
               dev_weights.Get(), dev_areas.Get(),
               trianglesCount, pointsCount, quadraturesOrder, dev_results.Get());

         break;
      }
      case AlgorythmGPU::Blocks:
      {
         /*dim3 dimBlock(QUADS_PER_BLOCK, POINTS_PER_BLOCK);
         dim3 dimGrid(1, pointsCount / POINTS_PER_BLOCK);*/

         dim3 dimBlock(POINTS_PER_BLOCK);
         dim3 dimGrid(pointsCount / POINTS_PER_BLOCK);

         laplace_solver_kernels::SolverKernelVector3sBlocks<<<
            dimGrid,
            dimBlock >>>(
               dev_quadPoints.Get(),
               dev_normals.Get(),
               dev_points.Get(),
               dev_weights.Get(), dev_areas.Get(),
               trianglesCount, pointsCount, quadraturesOrder, dev_results.Get());

         break;
      }
   }

   tryKernelLaunch();
   tryKernelSynchronize();
}

vector<float>& LaplaceSolverVector3s::GetResultGPU()
{
   dev_results.CopyToHost(results.data());
   return results;
}