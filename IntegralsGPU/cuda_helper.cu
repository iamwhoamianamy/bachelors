#include <stdio.h>
#include <cstdlib>
#include "cuda_helper.h"

void cuda_utilities::tryKernelLaunch()
{
   hipError_t cudaStatus = hipGetLastError();

   if(cudaStatus != hipError_t::hipSuccess)
   {
      printf("Kernel launch failed!\n");

      std::exit(0);
   }
}

void cuda_utilities::tryKernelSynchronize()
{
   hipError_t cudaStatus = hipDeviceSynchronize();

   if(cudaStatus != hipError_t::hipSuccess)
   {
      printf("Kernel synchrinisation failed!\n");

      std::exit(0);
   }
}