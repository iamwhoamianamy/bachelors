#include "hip/hip_runtime.h"
﻿#include "laplace_data.cuh"
#include <cmath>


__device__ __host__ inline real laplace_data::u(const real x, const real y, const real z)
{
   return 2 * x * x - y * y - z * z;
}

__device__ __host__ inline real laplace_data::gradUX(const real x, const real y, const real z)
{
   return 4 * x;
}

__device__ __host__ inline real laplace_data::gradUY(const real x, const real y, const real z)
{
   return -2 * y;
}

__device__ __host__ inline real laplace_data::gradUZ(const real x, const real y, const real z)
{
   return -2 * z;
}

__device__ float rsqrtf(float  x);
__device__ float rsqrtf(float  x);
__device__ double rsqrt(double  x);
__device__ float norm3df(float  a, float  b, float  c);
__device__ float rnorm3df(float  a, float  b, float  c);

__device__ real laplace_data::inverseDistanceGPU(const real x1, const real y1, const real z1,
                                               const real x2, const real y2, const real z2)
{
#ifdef REAL_IS_FLOAT
   return rsqrtf((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2));
   //return sqrtf((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2));
   //return norm3df(x1 - x2, y1 - y2, z1 - z2);
   //return rnorm3df(x1 - x2, y1 - y2, z1 - z2);
#endif // REAL_IS_FLOAT

#ifdef REAL_IS_DOUBLE
   return sqrt((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2));
#endif // REAL_IS_DOUBLE
}

__device__ real laplace_data::laplaceIntegral1GPU(const real qx, const real qy, const real qz,
                                                  const real px, const real py, const real pz,
                                                  const real nx, const real ny, const real nz)
{
   real dudnx = gradUX(qx, qy, qz) * nx;
   real dudny = gradUY(qx, qy, qz) * ny;
   real dudnz = gradUZ(qx, qy, qz) * nz;

   real l = inverseDistanceGPU(qx, qy, qz, px, py, pz);

   return (dudnx + dudny + dudnz) * l;
}

__device__ real laplace_data::laplaceIntegral2GPU(const real qx, const real qy, const real qz,
                                                  const real px, const real py, const real pz,
                                                  const real nx, const real ny, const real nz)
{
   real l = inverseDistanceGPU(qx, qy, qz, px, py, pz);

   real rx = nx * (px - qx);
   real ry = ny * (py - qy);
   real rz = nz * (pz - qz);

   return (rx + ry + rz) * u(qx, qy, qz) * (l * l * l);
}

__device__ real laplace_data::laplaceIntegral1GPU(const Vector3& q, const Vector3& p, const Vector3& n)
{
   return laplaceIntegral1GPU(q.x, q.y, q.z, p.x, p.y, p.z, n.x, n.y, n.z);
}

__device__ real laplace_data::laplaceIntegral2GPU(const Vector3& q, const Vector3& p, const Vector3& n)
{
   return laplaceIntegral2GPU(q.x, q.y, q.z, p.x, p.y, p.z, n.x, n.y, n.z);
}

__host__ real laplace_data::lengthBetweenCPU(const real x1, const real y1, const real z1,
                                             const real x2, const real y2, const real z2)
{
#ifdef REAL_IS_FLOAT
   return sqrtf((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2));
#endif // REAL_IS_FLOAT

#ifdef REAL_IS_DOUBLE
   return sqrt((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2) + (z1 - z2) * (z1 - z2));
#endif // REAL_IS_DOUBLE
}

__host__ real laplace_data::laplaceIntegral1CPU(const real qx, const real qy, const real qz,
                                                const real px, const real py, const real pz,
                                                const real nx, const real ny, const real nz)
{
   real dudnx = gradUX(qx, qy, qz) * nx;
   real dudny = gradUY(qx, qy, qz) * ny;
   real dudnz = gradUZ(qx, qy, qz) * nz;

   return (dudnx + dudny + dudnz) / lengthBetweenCPU(qx, qy, qz, px, py, pz);
}

__host__ real laplace_data::laplaceIntegral2CPU(const real qx, const real qy, const real qz,
                                                const real px, const real py, const real pz,
                                                const real nx, const real ny, const real nz)
{
   real l = lengthBetweenCPU(qx, qy, qz, px, py, pz);

   real rx = nx * (px - qx);
   real ry = ny * (py - qy);
   real rz = nz * (pz - qz);

   return (rx + ry + rz) * u(qx, qy, qz) / (l * l * l);
}

__host__ real laplace_data::laplaceIntegral1CPU(const Vector3& q, const Vector3& p, const Vector3& n)
{
   return laplaceIntegral1CPU(q.x, q.y, q.z, p.x, p.y, p.z, n.x, n.y, n.z);
}

__host__ real laplace_data::laplaceIntegral2CPU(const Vector3& q, const Vector3& p, const Vector3& n)
{
   return laplaceIntegral2CPU(q.x, q.y, q.z, p.x, p.y, p.z, n.x, n.y, n.z);
}