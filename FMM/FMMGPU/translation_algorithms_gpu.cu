#include "hip/hip_runtime.h"
#pragma once
#include "translation_algorithms.hpp"
#include <vector>
#include "kernels.cuh"
#include "cuda_helper.hpp"
#include "dev_ptr.hpp"
#pragma comment (lib, "cublas.lib")
#include "hipblaslt.h"

namespace kernels
{
   std::vector<real> addVectors(const std::vector<real>& a,
                                const std::vector<real>& b)
   {
      size_t size = a.size();

      cuda::DevPtr<real> dev_a(a.data(), a.size());
      cuda::DevPtr<real> dev_b(b.data(), b.size());
      cuda::DevPtr<real> dev_res(size);

      addingKernel<<<1, size>>>(dev_res.data(), dev_a.data(), dev_b.data());

      cuda::tryKernelLaunch();
      cuda::tryKernelSynchronize();

      real* res = new real[size];
      dev_res.copyToHost(res);

      return std::vector<real>(res, res + size);
   }

   void translateAllGPU(Vector3* result,
                        const real* a,
                        const Vector3* b,
                        size_t harmonicCount, size_t harmonicOrder)
   {
      size_t harmonicLength = (harmonicOrder + 1) * (harmonicOrder + 1);

      cuda::DevPtr<Vector3> result_dev(harmonicCount * harmonicLength);
      cuda::DevPtr<real> a_dev(a, harmonicCount * harmonicLength, 0);
      cuda::DevPtr<Vector3> b_dev(b, harmonicCount * harmonicLength, 0);

      dim3 BLOCKS((harmonicCount + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
      dim3 THREADS(THREADS_PER_BLOCK, harmonicOrder);

      kernels::translateAllGPUKernelSimpleXY<<<BLOCKS, THREADS>>>
         (result_dev.data(), a_dev.data(), b_dev.data(), harmonicCount, harmonicOrder);
      
      cuda::tryKernelLaunch();
      cuda::tryKernelSynchronize();

      result_dev.copyToHost(result);
   }

   void kernels::translateAllGPUMatrix(
      Complex* result,
      const Complex* a,
      const Complex* b,
      size_t harmonicCount,
      size_t harmonicOrder)
   {
      size_t harmonicLength = (harmonicOrder + 1) * (harmonicOrder + 1);

      size_t harLenPadded = math::nextDevisible(
         harmonicLength,
         THREADS_PER_BLOCK);

      size_t harCountPadded = math::nextDevisible(
         harmonicCount,
         THREADS_PER_BLOCK);

      cuda::DevPtr<Complex> aDev(a, harCountPadded * harLenPadded);
      ComplexKernelMatrix A;

      A.width = A.stride = harLenPadded;
      A.height = harCountPadded;
      A.elements = aDev.data();

      cuda::DevPtr<Complex> bDev(b, harLenPadded * harLenPadded);
      ComplexKernelMatrix B;

      B.width = B.stride = harLenPadded;
      B.height = harLenPadded;
      B.elements = bDev.data();

      cuda::DevPtr<Complex> cDev(harCountPadded * harLenPadded);
      ComplexKernelMatrix C;

      C.width = C.stride = harLenPadded;
      C.height = harCountPadded;
      C.elements = cDev.data();

      dim3 dimBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
      dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
      matMulKernel<<<dimGrid, dimBlock >>>(A, B, C);

      cuda::tryKernelLaunch();
      cuda::tryKernelSynchronize();

      cDev.copyToHost(result);
   }

   void translateAllGPUMatrixCuBLAS(
      hipComplex* result,
      const hipComplex* a,
      const hipComplex* b,
      size_t harmonicCount,
      size_t harmonicOrder)
   {
      size_t harmonicLength = (harmonicOrder + 1) * (harmonicOrder + 1);

      size_t harLenPadded = math::nextDevisible(
         harmonicLength,
         THREADS_PER_BLOCK);

      size_t harCountPadded = math::nextDevisible(
         harmonicCount,
         THREADS_PER_BLOCK);


      cuda::DevPtr<hipComplex> aDev(a, harCountPadded * harLenPadded);
      cuda::DevPtr<hipComplex> bDev(b, harLenPadded * harLenPadded);
      cuda::DevPtr<hipComplex> cDev(harCountPadded * harLenPadded);

      int m = harLenPadded;
      int k = harLenPadded;
      int n = harCountPadded;
      int lda = m, ldb = k, ldc = m;
      const hipComplex alf = make_hipComplex(1, 0);
      const hipComplex bet = make_hipComplex(0, 0);
      const hipComplex* alpha = &alf;
      const hipComplex* beta = &bet;
      
        // Create a handle for CUBLAS
      hipblasHandle_t handle;
      hipblasCreate(&handle);
      
       // Do the actual multiplication
      cublasCgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, 
                  bDev.data(), ldb, aDev.data(), lda, beta, cDev.data(), ldc);
      
       // Destroy the handle
      hipblasDestroy(handle);

      cDev.copyToHost(result);
   }

   size_t lmToIndex(int harmonicBegin,
                    int l, int m)
   {
      return harmonicBegin + l * l + l + m;
   }

   size_t lmToIndex(int l, int m)
   {
      return l * l + l + m;
   }

   __all__ real strangeFactor(int m, int mu)
   {
      return pow(-1, -0.5 * (abs(m) - abs(mu) - abs(m - mu)));
   }
}